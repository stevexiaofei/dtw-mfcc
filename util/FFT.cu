#include "hip/hip_runtime.h"
#include<assert.h>
#include<stdio.h>
#include<stdlib.h>
#include<cmath>

#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include<hip/hip_vector_types.h>

#include"FFT.h"
#include"util.h"
void CalcuCoes(int n,Complex* ptrCoes){
	for(i=0;i<n/2;i++){
		float temp=2Pai/float(n)*float(i)
		ptrCoes[i].real=cos(temp);
		ptrCoes[i].img=-sin(temp);
	}	
}

__global__ void cudaFFT(FtPat input, FtPat buffer, 
				Complex* Coes,int log_N){
	unsigned int nsample=input.batch;
	unsigned int length=input.length;

	Complex* data=input.data+sampleIndex*  ;
	Complex* bufferdata=buffer.data;
	Complex* ptrArray[2]={bufferdata,data};

	unsigned int sampleIndex=blockIdx.y*blockDim.y+threadIdx.y;
	unsigned int Index= blockIdx.x*blockDim.x+threadIdx.x;
	{
		unsigned int temp1=0;
		unsigned int temp2=0;
	    unsigned int exchangindex1=Index*2;
		unsigned int exchangindex2=Index*2+1;
		for(int i=0;i<log_N;i++){
	   		temp1=(temp1<<1)|(exchangindex1>>i&0x00000001);
			temp2=(temp2<<1)|(exchangindex2>>i&0x00000001);
		}
		bufferdata[temp1]=data[exchangindex1];
		bufferdata[temp2]=data[exchangindex2];
	}
    for(int i=0,unsigned int temp=0xFFFFFFFF;i<log_N;
					i++,temp=temp<<1)
	{
		unsigned int Stat=i%2,revStat=1-Stat;
		unsigned int indexa=((Index&temp)<<1)|(Index&~temp)
		unsigned int indexb=indexa|(1<<i);
		bufferfly(ptrArray[Stat][indexa],ptrArray[Stat][indexb],
						Coes[Index/(i+1)],ptrArray[revStat][indexa],
						ptrArray[revStat][indexb]);


	}	
}
void FFT(Ftpat in){

	
	assert(npoint%2==0);
    
	Complex* h_Coes=new Complex[in.length/2];
	CalcuCoes(npoint,Coes);
	Complex* d_Coes;
	size_t size=sizeof(Complex)*in.length/2;
	hipMalloc(&d_Coes, size);
	hipMemcpy(d_Coes, h_Coes, size, hipMemcpyHostToDevice);

	FtPat devFtPat=in,devFtPatBuffer=in;
    size_t size=sizeof(Complex)*in.length*in.nsample;
    hipMalloc(&devFtPat.data, size);
	cudaMelloc(&devFtPatBuffer.data,size);

	hipMemcpy(devFtPat.data, in.data ,size, 
					hipMemcpyHostToDevice);

    cudaFFT<<<  , >>>(devFtPat,devFtPatbuffer,d_Coes,N);



}
